
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_say_hello() {
  int N =
      gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
  printf("Hello world, CUDA! current thread%d  all thread %d\n", threadIdx.x,
         N);
}

int main() {
  printf("Hello world, CPU\n");
  cuda_say_hello<<<2, 3>>>();

  hipError_t cudaerr = hipDeviceSynchronize();
  if (cudaerr != hipSuccess)
    printf("kernel launch failed with error \"%s\".\n",
           hipGetErrorString(cudaerr));
  return 0;
}
